#include "comp_unit.cuh"
#include "cuda_ops.cuh"

const float alpha = 1.0f, beta = 0.0f;

ComputingUnit *ComputingUnit::instance = nullptr;
ComputingUnit &ComputingUnit::getInstance() {
    if (instance == nullptr)
        instance = new ComputingUnit();
    return *instance;
}

ComputingUnit::ComputingUnit() {
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        printf ("CUBLAS stat %u\n", stat);
        exit (EXIT_FAILURE);
    }
    hipdnnStatus_t err = hipdnnCreate(&cudnnHandle);
    if (err != HIPDNN_STATUS_SUCCESS) {
        std::cout << "Error occurred: " << err << std::endl;
        std::exit(EXIT_FAILURE);
    }
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
    auto cusparseStat = hipsparseCreate(&spHandle);
    assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);
}

CuMatrix ComputingUnit::wrapMatrix(Matrix m) {
    return CuMatrix(m, handle);
}

CuMatrix ComputingUnit::aggregate(CuMatrix &sparse, CuMatrix &dense) {
    CuMatrix C(Matrix(dense.getCols(), sparse.getRows(), (FeatType *) NULL), handle);

    hipsparseSpMatDescr_t desA;
    hipsparseDnMatDescr_t desB;
    hipsparseDnMatDescr_t desC;

    auto
    cusparseStat = hipsparseCreateCsr(&desA, sparse.getRows(), sparse.getCols(), sparse.nnz,
                                     sparse.csrRowPtr, sparse.csrColInd, sparse.csrVal,
                                     HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F
                                    );
    assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);
    cusparseStat = hipsparseCreateDnMat(&desB, dense.getCols(), dense.getRows(), dense.getCols(), dense.devPtr,
                                       HIP_R_32F, HIPSPARSE_ORDER_COL);
    assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);
    cusparseStat = hipsparseCreateDnMat(&desC, sparse.getRows(), dense.getCols(), sparse.getRows(), C.devPtr,
                                       HIP_R_32F, HIPSPARSE_ORDER_COL);
    assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);

    std::size_t buffer_size;
    cusparseStat = hipsparseSpMM_bufferSize(spHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
                                           &alpha, desA, desB, &beta,
                                           desC, HIP_R_32F, HIPSPARSE_MM_ALG_DEFAULT, &buffer_size
                                          );
    assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);
    float *buffer;
    hipMalloc ((void **)&buffer, buffer_size * sizeof(float));
    cusparseStat = hipsparseSpMM(spHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
                                &alpha, desA, desB, &beta,
                                desC, HIP_R_32F, HIPSPARSE_MM_ALG_DEFAULT, buffer
                               );
    assert(HIPSPARSE_STATUS_SUCCESS == cusparseStat);
    
    return C;
}

CuMatrix ComputingUnit::scaleRowsByVector(Matrix m, Matrix v) {
    CuMatrix cuM = wrapMatrix(m);
    CuMatrix cuV = wrapMatrix(v);
    thrust::device_ptr<float> m_ptr(cuM.devPtr);
    thrust::device_ptr<float> v_ptr(cuV.devPtr);
    thrust::transform(m_ptr, m_ptr + m.getNumElemts(),
                      thrust::make_permutation_iterator(
                          v_ptr,
                          thrust::make_transform_iterator(thrust::make_counting_iterator(0),
                                  linear_index_to_row_index<int>(m.getCols()))),
                      m_ptr,
                      thrust::multiplies<float>());
    return cuM;
}

CuMatrix ComputingUnit::hadamardSub(CuMatrix &matLeft, CuMatrix &matRight) {
    assert(matLeft.getRows() == matRight.getRows());
    assert(matLeft.getCols() == matRight.getCols());
    CuMatrix res(Matrix(matLeft.getRows(), matLeft.getCols(), (FeatType *)NULL), handle);

    thrust::device_ptr<float> cuLeft_ptr(matLeft.devPtr);
    thrust::device_ptr<float> cuRight_ptr(matRight.devPtr);
    thrust::device_ptr<float> res_ptr(res.devPtr);
    thrust::transform(cuLeft_ptr, cuLeft_ptr + matLeft.getNumElemts(),
                      cuRight_ptr,
                      res_ptr,
                      thrust::minus<float>());
    return res;
}

CuMatrix ComputingUnit::hadamardMul( CuMatrix &matLeft, CuMatrix &matRight) {
    assert(matLeft.getRows() == matRight.getRows());
    assert(matLeft.getCols() == matRight.getCols());
    CuMatrix res(Matrix(matLeft.getRows(), matLeft.getCols(), (FeatType *)NULL), handle);

    thrust::device_ptr<float> cuLeft_ptr(matLeft.devPtr);
    thrust::device_ptr<float> cuRight_ptr(matRight.devPtr);
    thrust::device_ptr<float> res_ptr(res.devPtr);

    thrust::transform(cuLeft_ptr, cuLeft_ptr + matLeft.getNumElemts(),
                      cuRight_ptr,
                      res_ptr,
                      thrust::multiplies<float>());

    return res;
}

CuMatrix ComputingUnit::softmaxRows( CuMatrix &mat) {
    CuMatrix res(Matrix(mat.getRows(), mat.getCols(), (FeatType *)NULL), handle);
    hipdnnTensorDescriptor_t srcTensorDesc, sftTensorDesc;
    hipdnnCreateTensorDescriptor(&srcTensorDesc);
    hipdnnCreateTensorDescriptor(&sftTensorDesc);
    hipdnnSetTensor4dDescriptor(srcTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                               mat.getRows(), 1, 1, mat.getCols());
    hipdnnSetTensor4dDescriptor(sftTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                               mat.getRows(), 1, 1, mat.getCols());
    hipdnnSoftmaxForward(cudnnHandle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_INSTANCE,
                        &alpha, srcTensorDesc, mat.devPtr,
                        &beta, sftTensorDesc, res.devPtr);
    return res;
}

CuMatrix ComputingUnit::activateBackward( CuMatrix &y, CuMatrix &gradient) {
    hipdnnActivationDescriptor_t actDesc;
    hipdnnCreateActivationDescriptor(&actDesc);
    hipdnnSetActivationDescriptor(actDesc, HIPDNN_ACTIVATION_TANH, HIPDNN_NOT_PROPAGATE_NAN, 1.0);

    hipdnnTensorDescriptor_t yDesc, dyDesc;
    hipdnnCreateTensorDescriptor(&yDesc);
    hipdnnSetTensor4dDescriptor(yDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                               y.getRows(), 1, 1, y.getCols());
    hipdnnCreateTensorDescriptor(&dyDesc);
    hipdnnSetTensor4dDescriptor(dyDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                               gradient.getRows(), 1, 1, gradient.getCols());

    hipdnnActivationBackward(cudnnHandle, actDesc,
                            &alpha, yDesc, y.devPtr,
                            dyDesc, gradient.devPtr,
                            yDesc, y.devPtr,
                            &beta, dyDesc, gradient.devPtr
                           );
    return gradient;
}


CuMatrix ComputingUnit::dot( Matrix &A, Matrix &B) {
    auto t0 = gtimers.getTimer("Dot Copy A");
    auto t1 = gtimers.getTimer("Dot Copy B");
    auto t2 = gtimers.getTimer("Dot Copy Out");
    t0->start();
    CuMatrix devA(A, handle);
    t0->stop();
    t1->start();
    CuMatrix devB(B, handle);
    t1->stop();
    CuMatrix devC = devA.dot(devB);
    t2->start();
    devC.updateMatrixFromGPU();
    t2->stop();
    return devC;
}

void ComputingUnit::activate(CuMatrix &A) {
    hipdnnTensorDescriptor_t srcTensorDesc;
    hipdnnCreateTensorDescriptor(&srcTensorDesc);
    hipdnnSetTensor4dDescriptor(srcTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                               A.getRows(), 1, 1, A.getCols());

    hipdnnActivationDescriptor_t actDesc;
    hipdnnCreateActivationDescriptor(&actDesc);
    hipdnnSetActivationDescriptor(actDesc, HIPDNN_ACTIVATION_TANH, HIPDNN_NOT_PROPAGATE_NAN, 1.0);
    hipdnnActivationForward(cudnnHandle, actDesc,
                           &alpha, srcTensorDesc, A.devPtr, &beta, srcTensorDesc, A.devPtr);
}

//** much slower than CPU only if Input Matrices are not loaded in GPU beforehand
unsigned ComputingUnit::checkAccuracy(CuMatrix &predictions, CuMatrix &labels) {

    unsigned rowSize = predictions.getCols();

    thrust::device_vector<FeatType *> row_starts(predictions.getRows());
    thrust::counting_iterator<int> idxfirst(0);

    thrust::transform(idxfirst, idxfirst + predictions.getRows(), row_starts.begin(),
                      setRowStarts(predictions.devPtr, rowSize));
    thrust::device_vector<unsigned> pred_results(predictions.getRows());
    thrust::transform(row_starts.begin(), row_starts.end(), pred_results.begin(),
                      findRowMaximum(rowSize));

    thrust::transform(idxfirst, idxfirst + predictions.getRows(), row_starts.begin(),
                      setRowStarts(labels.devPtr, rowSize));
    thrust::device_vector<unsigned> true_results(predictions.getRows());
    thrust::transform(pred_results.begin(), pred_results.end(), row_starts.begin(), true_results.begin(), isPredictCorrect(rowSize));

    unsigned totalCorrect = thrust::reduce(true_results.begin(), true_results.end(), (unsigned) 0, thrust::plus<unsigned>());
    return totalCorrect;
}

//** much slower than CPU only if Input Matrices are not loaded in GPU beforehand
float ComputingUnit::checkLoss(CuMatrix &preds, CuMatrix &labels) {
    unsigned rowSize = preds.getCols();

    thrust::counting_iterator<int> idxfirst(0);
    thrust::device_vector<FeatType *> row_starts(preds.getRows());
    thrust::transform(idxfirst, idxfirst + preds.getRows(), row_starts.begin(),
                      setRowStarts(labels.devPtr, rowSize));
    thrust::device_vector<unsigned> true_labels(preds.getRows());
    thrust::transform(row_starts.begin(), row_starts.end(), true_labels.begin(), findTrueLabel(rowSize));
    thrust::transform(idxfirst, idxfirst + preds.getRows(), row_starts.begin(),
                      setRowStarts(preds.devPtr, rowSize));
    thrust::device_vector<FeatType> losses(preds.getRows());
    thrust::transform(true_labels.begin(), true_labels.end(), row_starts.begin(), losses.begin(), getLoss(rowSize));
    float totalLoss = thrust::reduce(losses.begin(), losses.end(), (float) 0, thrust::plus<float>());
    return totalLoss;
}

