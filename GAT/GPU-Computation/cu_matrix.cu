#include "cu_matrix.cuh"

std::set<FeatType *> CuMatrix::MemoryPool;
CuMatrix::CuMatrix(Matrix M, const hipblasHandle_t &handle_)
    : Matrix(M.getRows(), M.getCols(), M.getData()) {
    cudaStat = hipError_t();
    handle = handle_;
    nnz = 0;
    csrVal = NULL;
    csrColInd = NULL;
    isSparse = 0;
    deviceMalloc();
    if (getData() != NULL) deviceSetMatrix();
}
CuMatrix::CuMatrix(){
    cudaStat = hipError_t();
    nnz = 0;
    csrVal = NULL;
    csrColInd = NULL;
    csrRowInd = NULL;
    isSparse = 0;
    setData(NULL);
};

Matrix CuMatrix::getMatrix() {
    updateMatrixFromGPU();
    return Matrix(getRows(), getCols(), getData());
}

void CuMatrix::freeGPU() {
    for (auto ptr : MemoryPool) hipFree(ptr);
}

CuMatrix CuMatrix::extractRow(unsigned row) {
    FeatType *data = getData() ? (getData() + row * getCols()) : NULL;
    CuMatrix rowVec;
    rowVec.handle = handle;
    rowVec.setData(data);
    rowVec.setRows(1);
    rowVec.setCols(getCols());
    rowVec.devPtr = devPtr + row * getCols();
    return rowVec;
}

void CuMatrix::deviceMalloc() {
    unsigned rows = this->getRows();
    unsigned cols = this->getCols();
    cudaStat = hipMalloc((void **)&devPtr, rows * cols * sizeof(FeatType));

    if (cudaStat != hipSuccess) {
        printf("device memory allocation failed %u\n", cudaStat);
        exit(EXIT_FAILURE);
    }
    MemoryPool.insert(devPtr);
}

void CuMatrix::deviceSetMatrix() {
    unsigned rows = this->getRows();
    unsigned cols = this->getCols();
    FeatType *data = this->getData();

    stat = hipblasSetMatrix(rows, cols, sizeof(float), data, rows, devPtr, rows);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        switch (stat) {
            case HIPBLAS_STATUS_NOT_INITIALIZED:
                printf("HIPBLAS_STATUS_NOT_INITIALIZED\n");
                break;
            case HIPBLAS_STATUS_INVALID_VALUE:
                printf("HIPBLAS_STATUS_INVALID_VALUE\n");
                break;
            case HIPBLAS_STATUS_MAPPING_ERROR:
                printf("HIPBLAS_STATUS_MAPPING_ERROR\n");
                break;
        }
        hipFree(devPtr);
        hipblasDestroy(handle);
        exit(EXIT_FAILURE);
    }
}

void CuMatrix::updateMatrixFromGPU() {
    unsigned rows = this->getRows();
    unsigned cols = this->getCols();
    if (getData() == NULL) setData(new FeatType[getNumElemts()]);
    FeatType *data = this->getData();

    stat = hipblasGetMatrix(rows, cols, sizeof(float), devPtr, rows, data, rows);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("data upload failed\n");
        switch (stat) {
            case HIPBLAS_STATUS_NOT_INITIALIZED:
                printf("HIPBLAS_STATUS_NOT_INITIALIZED\n");
                break;
            case HIPBLAS_STATUS_INVALID_VALUE:
                printf("HIPBLAS_STATUS_INVALID_VALUE\n");
                break;
            case HIPBLAS_STATUS_MAPPING_ERROR:
                printf("HIPBLAS_STATUS_MAPPING_ERROR\n");
                break;
        }
        hipFree(devPtr);
        hipblasDestroy(handle);
        exit(EXIT_FAILURE);
    }
}

CuMatrix::~CuMatrix() {
}

void CuMatrix::scale(const float &alpha) {
    hipblasSscal(handle, getNumElemts(), &alpha, devPtr, 1);
}

CuMatrix CuMatrix::dot(CuMatrix &B, bool A_trans, bool B_trans, float alpha,
                       float beta) {
    if (handle != B.handle) {
        std::cout << "Handle don't match\n";
        exit(EXIT_FAILURE);
    }
    hipblasOperation_t ATrans = A_trans ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t BTrans = B_trans ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    // 1. cublas is using col-major
    // 2. when cpy into/out device memory, it will do Transpose
    // 3. C=AB and C^T= (B^T*A^T)
    // This means just swap the order of multiplicaiton
    // Guide: https://peterwittek.com/cublas-matrix-c-style.html
    Matrix AT = Matrix(getCols(), getRows(), getData());
    Matrix BT = Matrix(B.getCols(), B.getRows(), B.getData());

    unsigned CRow = A_trans ? AT.getRows() : getRows();
    unsigned CCol = B_trans ? BT.getCols() : B.getCols();
    Matrix mat_C(CRow, CCol, (char *)NULL);  // real C

    unsigned k = A_trans ? getRows() : getCols();
    CuMatrix C(mat_C, handle);

    stat = hipblasSgemm(handle, BTrans, ATrans, C.getCols(), C.getRows(), k,
                       &alpha, B.devPtr, B.getCols(), devPtr, getCols(), &beta,
                       C.devPtr, C.getCols());
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("SGEMM ERROR\n");
        hipFree(devPtr);
        hipblasDestroy(handle);
        exit(EXIT_FAILURE);
    }
    return C;
}

CuMatrix CuMatrix::transpose() {
    // CuMatrix res(Matrix(getCols(), getRows(),
    // (char *)malloc(getNumElemts() * sizeof(FeatType))),
    //  handle);
    CuMatrix res(Matrix(getCols(), getRows(), (char *)NULL), handle);

    float alpha = 1.0;
    float beta = 0.;

    stat = hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, getRows(), getCols(),
                       &alpha, devPtr, getCols(), &beta, devPtr, getRows(),
                       res.devPtr, getRows());
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        hipblasDestroy(handle);
        exit(EXIT_FAILURE);
    }

    return res;
}
